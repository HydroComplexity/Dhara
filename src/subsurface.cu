#include "hip/hip_runtime.h"
/*
// Copyright (C) 2016, HydroComplexity Group
// All rights reserved.
//
// Distributed Hydrologicc and Regional Analysis (DHARA) Model
// DHARA model is made available as a restricted, non-exclusive, 
// non-transferable license for education and research purpose only, 
// and not for commercial use. See the LICENSE.txt for more details.
//
// Author: levuvietphong@gmail.com (Phong Le)
*/

#include "../include/main.h"
#include "../include/cusplib.h"
#include "../include/devconst.h"
#include "../include/global.h"


__device__ double maxcompssf (double a, double b)
{
    return (a < b) ? b : a;
}



__device__ void HydraulicConductivityAtInterface(double *knp1m, double *ke, double *kw, 
                double *kn, double *ks, double *ku, double *kd, int glob_ind, int i, int j, int k,
                int3 globsize)
{
    int sizex  = globsize.x;
    int sizey  = globsize.y;
    int sizez  = globsize.z;
    int sizexy = sizex * sizey;

    if ( k==0)
    {
        // top
        *kd = 0; //knp1m[glob_ind];
    } else {
        *kd = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind-sizexy]);
    }
    if (k==sizez-1 )
    {
        // bottom
        *ku = 0; //knp1m[glob_ind];
    } else {
        *ku = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind+sizexy]);
    }

    if ( j==0 )
    {
        // south
        *ks = 0; //knp1m[glob_ind];
    } else {
        *ks = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind-sizex]);
    }
    if ( j==sizey-1 )
    {
        // north
        *kn =0; // knp1m[glob_ind];
    } else {
        *kn = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind+sizex]);
    }

    if ( i==0 )
    {
        //  west
        *kw = 0; //knp1m[glob_ind];
    } else {
        *kw = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind-1]);
    }
    if ( i==sizex-1 )
    {
        // east 
        *ke = 0; // knp1m[glob_ind];
    } else {
        *ke = 0.5 * (knp1m[glob_ind] + knp1m[glob_ind+1]);
    }
}



__device__ void SetUpBlockMatrixSubsurface(double *a3d, double *thetanp1m, double *cnp1m, 
                double dx2inv, double dy2inv, double dz2inv, double ke, double kw, double kn,
                double ks, double ku, double kd, int tid, int glob_ind, int k, int3 globsize)
{
    int sizexyz = globsize.x * globsize.y * globsize.z;

    // lower 3 diagonals
    a3d[0*sizexyz+tid] = dz2inv * kd;
    a3d[1*sizexyz+tid] = dy2inv * ks;
    a3d[2*sizexyz+tid] = dx2inv * kw;

    // main diagonal
    a3d[3*sizexyz+tid] = Ss/dt * thetanp1m[glob_ind]/poros + cnp1m[glob_ind]/dt 
                         - (ke+kw) * dx2inv - (kn+ks) * dy2inv - (ku+kd) * dz2inv;
    
    // upper 3 diagonals
    a3d[4*sizexyz+tid] = dx2inv * ke;
    a3d[5*sizexyz+tid] = dy2inv * kn;
    a3d[6*sizexyz+tid] = dz2inv * ku;
}



__device__ void SetUpRightHandSideSubsurface(double *rhs, double *thetan, double *thetanp1m, 
                double *psinp1m, double *psin, double ku, double kd, double *cnp1m, double *tr,
                int *procmap, double *root, int tid, int i, int j, int k, int glob_ind, 
                int3 globsize)
{
    int id2d = j*globsize.x+i;
    int proc = procmap[id2d];
    double trodz;
    if (tr[id2d] > 0) {
        trodz = - tr[id2d] * root[proc*globsize.z+k] * sec_p_mm2dt_p_m / dz; // total amount during this time period       
    } 
    else {
        trodz = 0;
    }

    rhs[tid] = Ss/dt * thetanp1m[glob_ind]/poros * psin[glob_ind] + cnp1m[glob_ind]/dt * psinp1m[glob_ind] - (ku - kd)/dz + (thetan[glob_ind] - thetanp1m[glob_ind])/dt + trodz/dt;
}



__device__ void SetUpBoundaryConditionsSubsurface(double *a3d, double *rhs, int *bbc, int *tbc,
                int *sbc, int *nbc, int *wbc, int *ebc, double kd, double ku, double ks,
                double kn, double kw, double ke, double dx2inv, double dy2inv, double dz2inv,
                double *Psi_t, double *Psi_b, double *Psi_s, double *Psi_n, double *Psi_w,
                double *Psi_e, double *Knp1m, double *qw, double *qe,  double *qs, double *qn, double *qt,
                double *qb, int tid, int i, int j, int k, int3 globsize)
{
    int sizex   = globsize.x;
    int sizey   = globsize.y;
    int sizez   = globsize.z;
    int sizexyz = sizex * sizey * sizez;
    int bi, bj, bk;

    // Mapping to real boundary faces
    bi = k * sizey + j;
    bj = k * sizex + i;
    bk = j * sizex + i;

    // Top face boundary conditions
    if (tbc[bk] == 0)
    {   
        // Dirichlet Boundary
        if (k == 1)
        {   
            // Inner points
            a3d[0 * sizexyz + tid] = 0.0;
            rhs[tid] -= dz2inv * kd * Psi_t[bk];
        }

        if (k == 0)
        {   
            // Outer points
            for (int r=0; r<7; r++) 
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_t[bk];
        }
    }
    else 
    {  
        // Neumann Boundary
        if (k == 0) 
        {
            //a3d[3 * sizexyz + tid] += dz2inv * ku;
            //rhs[tid] += (dz - qt[bk]*dz/ku) * dz2inv * ku;
            rhs[tid] += (-qt[bk] * dz) * dz2inv;
        }
    }


    // Bottom face boundary conditions
    if (bbc[bk] == 0)
    {   
        // Dirichlet Boundary
        if (k == sizez-2)
        {
            a3d[6 * sizexyz + tid] = 0.0;
            rhs[tid] -= dz2inv * ku * Psi_b[bk];
        }
        if (k == sizez-1)
        {
            for (int r=0; r<7; r++)
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_b[bk];
        }
    }
    else 
    {  
        // Neumann Boundary
        if (k == sizez-1) 
        {
            //qb[bk] = kd;
            //a3d[3 * sizexyz + tid] += dz2inv * kd;
            //rhs[tid] += (-dz + qb[bk] * dz / kd) * dz2inv * kd;
            qb[bk] = Knp1m[tid];
            rhs[tid] += (qb[bk]*dz) * dz2inv;
        }
    }


    // South face boundary conditions
    if (sbc[bj] == 0)
    {   
        // Dirichlet Boundary
        if (j == 1)
        {   
            // Inner points
            a3d[1 * sizexyz + tid] = 0.0;
            rhs[tid] -= dy2inv * kn * Psi_s[bj];
        }

        if (j == 0)
        {   
            // Outer points
            for (int r=0; r<7; r++)
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_s[bj];
        }
    }
    else 
    {   
        // Neumann Boundary
        if (j == 0)
        {   
            //qs[bj] = ks;
            //a3d[3 * sizexyz + tid] += dy2inv * ks;
            //rhs[tid] += (-qs[bj] * dy / ks) * dy2inv * ks;
            rhs[tid] += (-qs[bj] * dy) * dy2inv;
        }
    }


    // North face boundary conditions
    if (nbc[bj] == 0) 
    {   
        // Dirichlet Boundary
        if (j == sizey-2)
        {   // Inner points
            a3d[5 * sizexyz + tid] = 0.0;
            rhs[tid] -= dy2inv * ks * Psi_n[bj];
        }

        if (j == sizey-1)
        {   
            // Outer points
            for (int r=0; r<7; r++)
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_n[bj];
        }
    } 
    else
    {   
        // Neumann Boundary
        if (j == sizey-1) 
        {
            //a3d[3 * sizexyz + tid] += dy2inv * kn;
            //rhs[tid] += (qn[bj] * dy / kn) * dy2inv * kn;
            rhs[tid] += (qn[bj] * dy) * dy2inv;
        }
    }

    
    // West face boundary conditions
    if (wbc[bi] == 0)
    {
        if (i == 1)
        {   
            // Inner points
            a3d[2 * sizexyz + tid] = 0.0;
            rhs[tid] -= dx2inv * ke * Psi_w[bi];
        }

        if (i == 0)
        {   
            // Outer points
            for (int r=0; r<7; r++) 
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_w[bi];
        }
    }
    else
    {
        if (i == 0)
        {
            //a3d[3 * sizexyz + tid] += dx2inv * kw;
            //rhs[tid] += (-qw[bi] * dx / kw) * dx2inv * kw;
            rhs[tid] += (-qw[bi] * dx) * dx2inv;
        }
    }


    // East face boundary conditions
    if (ebc[bi] == 0)
    {
        if (i == sizex-2)
        {   
            // Inner points
            a3d[4 * sizexyz + tid] = 0.0;
            rhs[tid] -= dx2inv * kw * Psi_e[bi];
        }

        if (i == sizex-1)
        {   
            // Outer points
            for (int r=0; r<7; r++) 
            {
                if (r == 3)
                    a3d[r * sizexyz + tid] = 1.0;
                else
                    a3d[r * sizexyz + tid] = 0.0;
            }
            rhs[tid] = Psi_e[bi];
        }
    }
    else
    {
        if (i == sizex-1)
        {
            //a3d[3 * sizexyz + tid] += dx2inv * ke;
            //rhs[tid] += (qe[bi] * dx / ke) * dx2inv * ke;
            rhs[tid] += (qe[bi] * dx) * dx2inv;
        }
    }
}


/**
 * @brief      Send transpiration from root process to entire domain on device
 *
 * @param      TR        Transpiration rate for entire domain
 * @param      TRroot    Transpiration collected at master process
 * @param      TRmap     Map of process on the domain
 * @param[in]  globsize  Size of the global domain
 */
__global__ void SendFluxDataToGrids(double *data, double *dataroot, int *procmap, int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizexy = globsize.x * globsize.y;    
    int ind;

    while (tid < sizexy)
    {
        ind = procmap[tid];
        data[tid] = dataroot[ind];

        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }    
}


__global__ void SetUpLinearSystemsSubsurface(double *a3d, double *rhs, double *psinp1m, 
                double *psin, double *thetanp1m, double *thetan, double *knp1m, double *cnp1m, 
                int *wbc, int *ebc, int *sbc, int *nbc, int *tbc, int *bbc, double *psi_w,
                double *psi_e, double *psi_s, double *psi_n, double *psi_t, double *psi_b,
                double *qw, double *qe, double *qs, double *qn, double *qt, double *qb, 
                double *tr, int *trmap, double *root, int3 globsize)
{
    int tid     = threadIdx.x + blockIdx.x * blockDim.x;
    int sizex   = globsize.x;
    int sizey   = globsize.y;
    int sizez   = globsize.z;
    int sizexy  = sizex * sizey;
    int sizexyz = sizex * sizey * sizez;

    int i, j, k, glob_ind;
    double dx2inv, dy2inv, dz2inv;
    double ke, kw, kn, ks, ku, kd;

    while (tid < sizexyz)
    {
        k = tid / sizexy;
        j = ( tid % sizexy ) / sizex;
        i = ( tid % sizexy ) % sizex;

        dx2inv = -1.0/(dx*dx);
        dy2inv = -1.0/(dy*dy);
        dz2inv = -1.0/(dz*dz);

        // Mapping to real 3D domain
        glob_ind = k * sizexy + j * sizex + i;

        HydraulicConductivityAtInterface(knp1m, &ke, &kw, &kn, &ks, &ku, &kd, glob_ind, i, j, k,
                                         globsize);

        SetUpBlockMatrixSubsurface(a3d, thetanp1m, cnp1m, dx2inv, dy2inv, dz2inv, ke, kw, kn, ks,
                                   ku, kd, tid, glob_ind, k, globsize);

        SetUpRightHandSideSubsurface(rhs, thetan, thetanp1m, psinp1m, psin, ku, kd, cnp1m, tr,
                                     trmap, root, tid, i, j, k, glob_ind, globsize);

        SetUpBoundaryConditionsSubsurface(a3d, rhs, bbc, tbc, sbc, nbc, wbc, ebc, 
                                          kd, ku, ks, kn, kw, ke, dx2inv, dy2inv, dz2inv,
                                          psi_t, psi_b, psi_s, psi_n, psi_w, psi_e, knp1m,
                                          qw, qe,  qs, qn, qt, qb, tid, i, j, k, globsize);

        __syncthreads();    // All thread must sync at this point
        tid += blockDim.x * gridDim.x;
    }
}


__global__ void GetIterationDifference( double *psinp1m, double *psinp1mp1, double* deltam, 
                int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizexyz = globsize.x * globsize.y * globsize.z;

    while (tid < sizexyz){
        // Shuold be percentage difference
        //deltam[tid] = abs(psinp1mp1[tid] - psinp1m[tid]);
        deltam[tid] = abs((psinp1mp1[tid] - psinp1m[tid])/psinp1mp1[tid]);        
        
        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }
}



__global__ void ModifiedPicardUpdate(double *psinp1m, double *psinp1mp1, int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizexyz = globsize.x * globsize.y * globsize.z;    

    while (tid < sizexyz){
        psinp1m[tid] = am * psinp1m[tid] + (1-am) * psinp1mp1[tid];

        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }
}



__global__ void SubsurfaceEstimateInfiltrationPonding(double *psinp1mp1, double *knp1m, 
                double *qt, double *qss, double *psi_top, double *ph, int *tbc, double *hpoten,
                int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizex = globsize.x;
    int sizey = globsize.y;

    while (tid < sizex * sizey) {
        if (tbc[tid] == 1) 
        {  
            // Flux boundary
            qss[tid] = qt[tid];
            // surface poinding is not zero.
            //ph[tid]  = 0.0;
            ph[tid] = maxcompssf(hpoten[tid] - qss[tid] * dt, 0.0);
        }
        else
        {
            // Pressure head boundary
            qss[tid] = -knp1m[tid] * (psinp1mp1[tid] - psi_top[tid] - dz) / dz;
            ph[tid]  = maxcompssf(psi_top[tid] - qss[tid]*dt, 0.0);
        }

        tid += blockDim.x * gridDim.x;
    }
}


__global__ void WaterFluxEstimate (double *knp1m, double *psinp1mp1, double *psin, double *thetanp1mp1, double *thetan, double *E_soil, double *TR, double *qss,
                double *bcqw, double *bcqe, double *bcqs, double *bcqn, double *bcqt, double *bcqb,
                double *quflux, double *qdflux, double *qwflux, double *qeflux, double *qsflux, double *qnflux,
                double *dtheta, double *transp, double *evapo, double *ssflux, 
                int *wbc, int *ebc, int *sbc, int *nbc, int *tbc, int *bbc,
                int *procmap, double *root, int3 globsize)
{
    int tid     = threadIdx.x + blockIdx.x * blockDim.x;
    int sizex   = globsize.x;
    int sizey   = globsize.y;
    int sizez   = globsize.z;
    int sizexy  = sizex * sizey;
    int sizexyz = sizex * sizey * sizez;
    int i, j, k, id2d, glob_ind;
    int N = globsize.x;
    int M = globsize.y;
    int P = globsize.z;
    double ke, kw, kn, ks, ku, kd;
    int proc;
    int bi, bj, bk;


    while (tid < sizexyz){
        k = tid / sizexy;
        j = ( tid % sizexy ) / sizex;
        i = ( tid % sizexy ) % sizex;         

        // Mapping to real 3D domain
        glob_ind = k * sizexy + j * sizex + i;

        // Maping to real 2D doamin
        id2d = j*globsize.x+i;
        proc = procmap[id2d];

        // Mapping to real boundary faces
        bi = k * sizey + j;
        bj = k * sizex + i;
        bk = j * sizex + i;

        // Calculate Hydraulic conductivity at interfaces
        HydraulicConductivityAtInterface(knp1m, &ke, &kw, &kn, &ks, &ku, &kd, glob_ind, i, j, k,
                                         globsize);


        // 1. Estimate water flux: Down, east, and north-ward positive.
        // Initialization
        qdflux[glob_ind] = 0; 
        quflux[glob_ind] = 0;
        qwflux[glob_ind] = 0;
        qeflux[glob_ind] = 0;
        qsflux[glob_ind] = 0;
        qnflux[glob_ind] = 0;        

        if (k != 0){
            qdflux[glob_ind] = kd * ((psinp1mp1[glob_ind - M*N] - psinp1mp1[glob_ind]) / dz + 1); // [m/dtime]
        }
        if (k != P - 1){
            quflux[glob_ind] = ku * ((psinp1mp1[glob_ind] - psinp1mp1[glob_ind + M*N]) / dz + 1);

            if (bbc[bk] == 0)
                quflux[glob_ind + M*N] = ku * ((psinp1mp1[glob_ind] - psinp1mp1[glob_ind + M*N]) / dz + 1);
        }

        if (i != 0){
            qwflux[glob_ind] = kw * (psinp1mp1[glob_ind - 1] - psinp1mp1[glob_ind]) / dx;

            if (wbc[bi] == 0)
                qwflux[glob_ind-1] = kw * (psinp1mp1[glob_ind - 1] - psinp1mp1[glob_ind]) / dx;
        }
        if (i != N - 1){
            qeflux[glob_ind] = ke * (psinp1mp1[glob_ind] - psinp1mp1[glob_ind + 1]) / dx;

            if (ebc[bi] == 0)
                qeflux[glob_ind + 1] = ke * (psinp1mp1[glob_ind] - psinp1mp1[glob_ind + 1]) / dx;
        }

        if (j != 0){
            qsflux[glob_ind] = ks * (psinp1mp1[glob_ind - N] - psinp1mp1[glob_ind]) / dy;

            if (sbc[bj] == 0)
                qsflux[glob_ind - N] = ks * (psinp1mp1[glob_ind - N] - psinp1mp1[glob_ind]) / dy;
        }
        if (j != M - 1) {
            qnflux[glob_ind] = kn * (psinp1mp1[glob_ind] - psinp1mp1[glob_ind + N]) / dy;

            if (nbc[bj] == 0)
                qnflux[glob_ind + N] = kn * (psinp1mp1[glob_ind] - psinp1mp1[glob_ind + N]) / dy;
        }

        //Boundary condition: Only Neumann Boundary works & free bottom flow
        if (k == 0){
            qdflux[glob_ind] = bcqt[bk]; //qss[glob_ind];
        }
        if (k == P - 1){
            if (bbc[bk] != 0)
                quflux[glob_ind] = bcqb[bk];
        }

        if (i == N - 1){
            if (ebc[bi] != 0)
                qeflux[glob_ind] = bcqe[bi];
        }
        if (i == 0){
            if (wbc[bi] != 0)
                qwflux[glob_ind] = bcqw[bi];
        }

        if (j == M - 1) {
            if (nbc[bj] != 0)
                qnflux[glob_ind] = bcqn[bj];
        }
        if (j == 0){
            if (sbc[bj] != 0)
                qsflux[glob_ind] = bcqs[bj];
        }

        // 2. Diff in soil moisutre
        dtheta[glob_ind] = thetanp1mp1[glob_ind] - thetan[glob_ind]; // [-]

        // 3. Get transpiration
        if (TR[id2d] > 0){
            transp[glob_ind] = - TR[id2d] * sec_p_mm2dt_p_m * root[proc*globsize.z+k]; // [m] during the time step            
        }
        else {
            transp[glob_ind] = 0;            
        }


        // 4. Soil storage changes
        ssflux[glob_ind] = (psinp1mp1[glob_ind] - psin[glob_ind]) *(Ss / dt * thetanp1mp1[glob_ind] / poros) * dz; //[m/dtime]


        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }
}


void GatherFluxesDomain(ProjectClass *project, VerticalCanopyClass *vertcanopies,
                        VerticalSoilClass *vertsoils, SubsurfaceFlowClass *subsurface_host, 
                        SubsurfaceFlowClass *subsurface_dev, int rank, int procsize, int3 globsize,
                        int3 domsize, int2 topolsize, int2 topolindex, MPI_Comm *cartComm)
{
    int isroot = rank == MPI_MASTER_RANK;

    MPI_Gather(vertcanopies->TR_can, 1, MPI_DOUBLE, subsurface_host->TR_root, 1, MPI_DOUBLE, 0, *cartComm);
    MPI_Gather(vertsoils->ppt_ground, 1, MPI_DOUBLE, subsurface_host->ppt_root, 1, MPI_DOUBLE, 0, *cartComm);
    MPI_Gather(vertsoils->E_soil, 1, MPI_DOUBLE, subsurface_host->E_soil_root, 1, MPI_DOUBLE, 0, *cartComm);

    if (isroot)
    {
        SafeCudaCall( hipMemcpy(subsurface_dev->TR_root, subsurface_host->TR_root, 
                      procsize*sizeof(double), hipMemcpyHostToDevice) );
        SendFluxDataToGrids<<<TSZ,BSZ>>>(subsurface_dev->TR, subsurface_dev->TR_root,
                                         subsurface_dev->procmap, globsize);

        SafeCudaCall( hipMemcpy(subsurface_dev->ppt_root, subsurface_host->ppt_root, 
                      procsize*sizeof(double), hipMemcpyHostToDevice) );        
        SendFluxDataToGrids<<<TSZ,BSZ>>>(subsurface_dev->ppt_ground, subsurface_dev->ppt_root,
                                         subsurface_dev->procmap, globsize);

        SafeCudaCall( hipMemcpy(subsurface_dev->E_soil_root, subsurface_host->E_soil_root, 
                      procsize*sizeof(double), hipMemcpyHostToDevice) );        
        SendFluxDataToGrids<<<TSZ,BSZ>>>(subsurface_dev->E_soil, subsurface_dev->E_soil_root,
                                         subsurface_dev->procmap, globsize);
        cudaCheckError("SendFluxDataToGrids");
    }

}



/**
 * @brief      Run the subsurface flow model in device
 *
 * @param      timeforcings     Class including time forcings info
 * @param      overland_host    Overland flow class in host memory
 * @param      overland_dev     Overland flow class in device memory
 * @param      subsurface_host  Subsurface flow class in host memory
 * @param      subsurface_dev   Subsurface flow class in device memory
 * @param      a3d_cusp         Left hand side matrix A in cusp format
 * @param      psinp1mp1_cusp   Pressure head at n+1,m+1 in cusp format
 * @param      rhs3d_cusp       Right hand side vector b in cusp format
 * @param      id3d             The identity for linear system solver
 * @param      deltam_cusp      The difference between 2 iters in cusp format
 * @param      maxError         The maximum error of vector difference
 * @param[in]  rank             Global rank of the current MPI process
 * @param[in]  procsize         Total number of MPI processes available
 * @param[in]  globsize         Size of the global domain
 * @param[in]  t                Current time step running
 * @param[in]  num_steps        The number steps for simulation
 */
void SubsurfaceFlowModel(TimeForcingClass * &timeforcings, OverlandFlowClass * &overland_host,
                         OverlandFlowClass * &overland_dev, SubsurfaceFlowClass * &subsurface_host,
                         SubsurfaceFlowClass * &subsurface_dev, cuspdev_diamat &a3d_cusp,
                         cuspdev_1d &psinp1mp1_cusp, cuspdev_1d &rhs3d_cusp, cuspdev_idoper &id3d,
                         cuspdev_1d &deltam_cusp, thrustdev_iter &maxError, 
                         thrustdev &quflux_thrust, thrustdev &qdflux_thrust, thrustdev &qwflux_thrust, thrustdev &qeflux_thrust, thrustdev &qsflux_thrust, thrustdev &qnflux_thrust,
                         thrustdev &dtheta_thrust, thrustdev &transp_thrust, thrustdev &evapo_thrust, thrustdev &ssflux_thrust,
                         int rank, int procsize,
                         int3 globsize, int t, int num_steps)
{
    int sizexy  = globsize.x * globsize.y;
    int sizexyz = globsize.x * globsize.y * globsize.z;
    int maxiter = subsurface_host->maxiter;
    int picardmethod = subsurface_host->picardmethod;

    int runflag, niter;
    double stop_tol;

    SafeCudaCall( hipMemcpy(subsurface_dev->psinp1m, subsurface_dev->psin,
                             sizexyz*sizeof(double), hipMemcpyDeviceToDevice) );

    runflag = 0;
    niter = 0;

    EstimateFluxes<<<TSZ,BSZ>>>(overland_dev->ph, overland_dev->hpoten, overland_dev->qcapa,
                  subsurface_dev->psinp1m, subsurface_dev->knp1m, subsurface_dev->ppt_ground, 
                  subsurface_dev->E_soil, subsurface_dev->ksat, globsize);
    cudaCheckError("EstimateFluxes");

    while (runflag == 0 && niter < maxiter)
    {
        // Convert pressure head (psi) to moisture (theta)
        vanGenuchten<<<TSZ,BSZ>>>(subsurface_dev->cnp1m, subsurface_dev->thetanp1m,
                    subsurface_dev->ksat, subsurface_dev->knp1m, subsurface_dev->psinp1m,
                    globsize );
        cudaCheckError("vanGenuchten");

        // Boundary switching
        IdentifyTopBoundary<<<TSZ,BSZ>>>(overland_dev->hpoten, overland_dev->qcapa,
                           subsurface_dev->bct, subsurface_dev->bcqt,
                           subsurface_dev->psinp1m, subsurface_dev->bcpsit,
                           subsurface_dev->thetan, subsurface_dev->ksat,
                           globsize );
        cudaCheckError("IdentifyTopBoundary");

        // Set A, b, and boundary conditions
        SetUpLinearSystemsSubsurface<<<TSZ, BSZ>>>(subsurface_dev->a3d, subsurface_dev->rhs3d,
                                    subsurface_dev->psinp1m, subsurface_dev->psin,
                                    subsurface_dev->thetanp1m, subsurface_dev->thetan,
                                    subsurface_dev->knp1m, subsurface_dev->cnp1m,
                                    subsurface_dev->bcw, subsurface_dev->bce,
                                    subsurface_dev->bcs, subsurface_dev->bcn,
                                    subsurface_dev->bct,subsurface_dev->bcb,
                                    subsurface_dev->bcpsiw, subsurface_dev->bcpsie,
                                    subsurface_dev->bcpsis, subsurface_dev->bcpsin,
                                    subsurface_dev->bcpsit, subsurface_dev->bcpsib,
                                    subsurface_dev->bcqw, subsurface_dev->bcqe,
                                    subsurface_dev->bcqs, subsurface_dev->bcqn,
                                    subsurface_dev->bcqt, subsurface_dev->bcqb, 
                                    subsurface_dev->TR, subsurface_dev->procmap, 
                                    subsurface_dev->rda, globsize );
        cudaCheckError("SetUpLinearSystemsSubsurface");

        // Solve linear systems
        cusp::monitor <double> monitor(rhs3d_cusp, 100, 1e-8);
        cusp::krylov::bicgstab(a3d_cusp, psinp1mp1_cusp, rhs3d_cusp, monitor, id3d);

        // Again, convert psi to theta
        vanGenuchten<<<TSZ,BSZ>>>(subsurface_dev->cnp1m, subsurface_dev->thetanp1mp1,
                                  subsurface_dev->ksat, subsurface_dev->knp1m,
                                  subsurface_dev->psinp1mp1, globsize );
        cudaCheckError("vanGenuchten");

        niter += 1;

        // Get the difference between 2 iterations and find maxError
        if (picardmethod == 0)
        {   
            // use pressure as primary
            GetIterationDifference<<<TSZ,BSZ>>>(subsurface_dev->psinp1m, subsurface_dev->psinp1mp1,
                                  subsurface_dev->deltam, globsize );
            cudaCheckError("GetIterationDifference_psi");
            stop_tol = subsurface_host->tolerance_psi;
        } else {
            // use moisture as primary
            GetIterationDifference<<<TSZ,BSZ>>>(subsurface_dev->thetanp1m,
                                  subsurface_dev->thetanp1mp1, subsurface_dev->deltam, globsize );
            cudaCheckError("GetIterationDifference_theta");
            stop_tol = subsurface_host->tolerance_theta;
        }

        maxError = thrust::max_element(deltam_cusp.begin(), deltam_cusp.end());

        // check the maximum error to test convergence
        if (*maxError < stop_tol)
        {
            // converged
            runflag = 1;
        } else {
            // not convereged yet, update and repeat
            ModifiedPicardUpdate<<<TSZ,BSZ>>>(subsurface_dev->psinp1m, subsurface_dev->psinp1mp1,
                                globsize);
            cudaCheckError("ModifiedPicardUpdate");
        }
    }

    // Estimate infiltration to soil
    SubsurfaceEstimateInfiltrationPonding<<<TSZ,BSZ>>>(subsurface_dev->psinp1mp1, 
                                         subsurface_dev->knp1m, subsurface_dev->bcqt,
                                         subsurface_dev->qss, subsurface_dev->bcpsit, 
                                         overland_dev->ph, subsurface_dev->bct, overland_dev->hpoten, 
                                         globsize);

    WaterFluxEstimate<<<TSZ,BSZ>>>(subsurface_dev->knp1m, subsurface_dev->psinp1mp1, subsurface_dev->psin, subsurface_dev->thetanp1mp1, subsurface_dev->thetan, subsurface_dev->E_soil, subsurface_dev->TR, subsurface_dev->qss,
                     subsurface_dev->bcqw, subsurface_dev->bcqe, subsurface_dev->bcqs, subsurface_dev->bcqn, subsurface_dev->bcqt, subsurface_dev->bcqb, 
                     subsurface_dev->quflux, subsurface_dev->qdflux, subsurface_dev->qwflux, subsurface_dev->qeflux, subsurface_dev->qsflux, subsurface_dev->qnflux, 
                     subsurface_dev->dtheta, subsurface_dev->transp, subsurface_dev->evapo, subsurface_dev->ssflux, 
                     subsurface_dev->bcw, subsurface_dev->bce,
                     subsurface_dev->bcs, subsurface_dev->bcn,
                     subsurface_dev->bct, subsurface_dev->bcb,
                     subsurface_dev->procmap, subsurface_dev->rda, globsize);

    // Check the mass blance
    double sum_qu, sum_qd, sum_qw, sum_qe, sum_qs, sum_qn;
    double sum_ssflux, sum_tr, sum_dtheta;
    double mb_subWater;
    
    sum_qu = thrust::reduce(quflux_thrust.begin(), quflux_thrust.end());
    sum_qd = thrust::reduce(qdflux_thrust.begin(), qdflux_thrust.end());
    sum_qw = thrust::reduce(qwflux_thrust.begin(), qwflux_thrust.end());
    sum_qe = thrust::reduce(qeflux_thrust.begin(), qeflux_thrust.end());
    sum_qs = thrust::reduce(qsflux_thrust.begin(), qsflux_thrust.end());
    sum_qn = thrust::reduce(qnflux_thrust.begin(), qnflux_thrust.end());

    sum_ssflux = thrust::reduce(ssflux_thrust.begin(), ssflux_thrust.end());
    sum_tr = thrust::reduce(transp_thrust.begin(), transp_thrust.end());
    sum_dtheta = thrust::reduce(dtheta_thrust.begin(), dtheta_thrust.end());

    // Estimate mass balance for the subsurface soil water
    mb_subWater = (sum_dtheta)* subsurface_host->dz + ((sum_qe - sum_qw) + (sum_qn - sum_qs) + (sum_qu - sum_qd)) * subsurface_host->dt - sum_tr  + sum_ssflux * subsurface_host->dt; // [m]
    subsurface_host->mb_subsurfaceW [t] = mb_subWater;
    //printf("Subsurface water balance = %f \n",mb_subWater);



    cudaCheckError("SubsurfaceEstimateInfiltrationPonding");

    SafeCudaCall( hipMemcpy(overland_dev->waterdepth, overland_dev->ph, 
                             sizexy*sizeof(double), hipMemcpyDeviceToDevice) );

    SafeCudaCall( hipMemcpy(subsurface_dev->psin, subsurface_dev->psinp1mp1,
                             sizexyz*sizeof(double), hipMemcpyDeviceToDevice) );

    SafeCudaCall( hipMemcpy(subsurface_dev->thetan, subsurface_dev->thetanp1m, 
                             sizexyz*sizeof(double), hipMemcpyDeviceToDevice) );

    SafeCudaCall( hipMemcpy(subsurface_dev->psiout, subsurface_dev->psin, 
                             sizexyz*sizeof(double), hipMemcpyDeviceToHost) );

    SafeCudaCall( hipMemcpy(subsurface_dev->thetaout, subsurface_dev->thetan, 
                             sizexyz*sizeof(double), hipMemcpyDeviceToHost) );
}
